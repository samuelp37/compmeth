#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void add_simple(int *a, int *b, int *c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

extern void cuda_computeAddition(int *a, int *b, int *c,int N)
{
	int size = N*sizeof(int);
	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add_simple<<<N,1>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a); hipFree(d_b); hipFree(d_c);

}
#include <hip/hip_runtime.h>

#include <stdio.h>


extern void cuda_upload(int *a, int N){

	int size = N*sizeof(int);
	int *d_a;
	hipMalloc((void **)&d_a, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipFree(d_a);

}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include ""

__global__ void add_simple(int *a, int *b, int *c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void add_advanced(int *a, int *b, int *c,int N)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < N){
		c[index] = a[index] + b[index];
	}
}

extern void cuda_computeAddition(int *a, int *b, int *c,int N)
{
	int size = N*sizeof(int);
	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add_simple<<<N,1>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a); hipFree(d_b); hipFree(d_c);

}

extern void cuda_computeAddition_advanced(int *a, int *b, int *c, int N,int M)
{
	int size = N*sizeof(int);
	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add_advanced <<<N/M+1, M >> >(d_a, d_b, d_c,N);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a); hipFree(d_b); hipFree(d_c);

}